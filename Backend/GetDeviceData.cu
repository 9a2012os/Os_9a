#include "hip/hip_runtime.h"
#include "GetDeviceData.h"
#include <hip/hip_runtime.h>

DeviceData GetDeviceInformation()
{
	// Device properties.
	DeviceData gpuData;
	hipDeviceProp_t gpuDeviceProperties;
	int deviceCount,i;
	
	// Getting devices count.
	hipGetDeviceCount(&deviceCount);
	
	for(i=0;i<deviceCount;i++)
	{
		// Initializing gpuData variable.
		gpuData = (struct deviceData*)malloc(sizeof(struct deviceData));

		// Getting the device data.
		
		// Getting device name.
		hipGetDeviceProperties(&gpuDeviceProperties, i);
		gpuData->deviceName = (char*)malloc(sizeof(char)*strlen(gpuDeviceProperties.name));
		strcpy(gpuData->deviceName,gpuDeviceProperties.name);
		//printf("name:: %s\n",gpuData->deviceName);
		
		// Getting device clock rate.
		gpuData->clockRate=gpuDeviceProperties.clockRate;
		
		// Getting device memory information.
		gpuData->totalMemory = (gpuDeviceProperties.totalGlobalMem)/1000;
		gpuData->registersPerMp= gpuDeviceProperties.regsPerBlock;		
		gpuData->threadsPerBlock= gpuDeviceProperties.maxThreadsPerBlock;
		gpuData->threadsInWrap= gpuDeviceProperties.warpSize;
		gpuData->mpCount=gpuDeviceProperties.multiProcessorCount;
		gpuData->sharedMemoryPerMp =gpuDeviceProperties.sharedMemPerBlock;
		gpuData->threadsPerBlock = gpuDeviceProperties.maxThreadsPerBlock;

	}
	return gpuData;
}
